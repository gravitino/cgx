#include "hip/hip_runtime.h"
#include "cgx/cgx.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
using namespace cgx;

__global__ void kernel () {

    // get the current grid
    const auto group = this_grid();

    // alternatively you can use the current block
    // or any coalesced group within a block
    // more general: any object that exposed the member functions
    // thread_rank() and size()
    // const auto group = this_thread_block();
    // const auto group = tiled_partition<2>(this_thread_block());

    // the whole group generates scalar (1D) indices
    // between 0 and 7 (exlusive) with unit step size
    // 0, 1, 2, 3, 4, 5, 6, 7
    for (auto dim : range(group, 7))
        printf("%ld %ld %ld\n", dim.x, dim.y, dim.z);


    // the whole group generates sclar (1D) indices
    // from -3 to 4 with step size 2 (think about Python's range)
    // -3, -2, -1, 0, 1, 2, 3
    //for (auto dim : range(group, -3, 4, 2))
    //    printf("%ld %ld %ld\n", dim.x, dim.y, dim.z);

    // the whole group generates all 3D indices from the
    // Cartesian product [0,1) x [0, 2) x [0, 3) with unit step size
    //for (auto dim : range(group, dim3_t(1, 2, 3)))
    //    printf("%ld %ld %ld\n", dim.x, dim.y, dim.z);

    // the whole group generates all 3D indices from the
    // Cartesian product [-1,3) x [-2, 2) x [-3, 2) with
    // custom step sizes 1, 2, 3 for each dimension
    //for (auto dim : range(group, dim3_t(-1,-2,-3),
    //                             dim3_t( 3, 2, 1),
    //                             dim3_t( 1, 2, 3)))
    //    printf("%ld %ld %ld\n", dim.x, dim.y, dim.z);
}

int main (int argc, char * argv[]) {

    void * args[0];
    dim3 blocks (2, 1, 1);
    dim3 threads(2, 2, 1);

    // launch the kernel with any suitable configuration (1D, 2D, 3D)
    // whatever fits your needs best. index generation is accomlished
    // with the range iterator
    hipLaunchCooperativeKernel((void*)kernel, blocks, threads, args, 0);
    hipDeviceSynchronize();
}
